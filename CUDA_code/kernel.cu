
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <omp.h>
#include <math.h>       /* fabsf */
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define DEBUG 0
using namespace std;
//Error check-----
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
//Error check-----
//This is a very good idea to wrap your calls with that function.. Otherwise you will not be able to see what is the error.
//Moreover, you may also want to look at how to use cuda-memcheck and cuda-gdb for debugging.


__global__ void updater(double* rv, double* cv,int* adj, int* xadj,int temp)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<temp){
    int st = xadj[i+1];
    int ed = xadj[i];
    double rsum = 0;
    for(int k = ed;k<st;k++){
      rsum += cv[adj[k]];
    }
    rv[i]= 1/rsum;
    }
}
__global__ void updaters(double* rv, double* cv,int* tadj, int* txadj,int temp)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<temp){
    int kt = txadj[i+1];
    int kd = txadj[i];
    double csum = 0;
    for(int m = kd;m<kt;m++){
      csum += rv[tadj[m]];
    }
    cv[i]=1/csum;   
    }
}

__global__ void updatere(double* rv, double* cv,int* adj, int* xadj,double* max,int temp)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i< temp){
    int st = xadj[i+1];
    int ed = xadj[i];
   
    double total = 0;
    for(int k = ed;k<st;k++){
        total += cv[adj[k]]* rv[i];
    }
    total = fabs(1-total);
    if(*max < total){
      *max = total;
    }
    }
}


void wrapper(int* adj, int* xadj, int* tadj, int* txadj, double* rv, double* cv, int* nov, int* nnz, int siter){
  
  printf("Wrapper here! \n");
  
  //TO DO: DRIVER CODE
    int* adj_d, *xadj_d, *tadj_d, *txadj_d;
    gpuErrchk(hipMalloc( (void**) &adj_d, (*nnz) * sizeof(int)));
    gpuErrchk(hipMemcpy(adj_d, adj, (*nnz) * sizeof(int), hipMemcpyHostToDevice ));

    gpuErrchk(hipMalloc( (void**) &xadj_d, (*nov) * sizeof(int)));
    gpuErrchk(hipMemcpy(xadj_d, xadj, (*nov) * sizeof(int), hipMemcpyHostToDevice ));

    gpuErrchk(hipMalloc( (void**) &tadj_d, (*nnz) * sizeof(int)));
    gpuErrchk(hipMemcpy(tadj_d, tadj,(*nnz) * sizeof(int), hipMemcpyHostToDevice ));

    gpuErrchk(hipMalloc( (void**) &txadj_d, (*nov) * sizeof(int)));
    gpuErrchk(hipMemcpy(txadj_d, txadj,(*nov) * sizeof(int), hipMemcpyHostToDevice ));

   for(int i = 0;i<*nov;i++){
        rv[i]=1;
        cv[i]=1;
    }
    hipEvent_t start, stop;
    double* rv_d;
    gpuErrchk(hipMalloc((void **)&rv_d, (*nov) * sizeof(double)));
    gpuErrchk(hipMemcpy(rv_d, rv, (*nov) * sizeof(double), hipMemcpyHostToDevice));
    double* cv_d;
    gpuErrchk(hipMalloc((void **)&cv_d, (*nov) * sizeof(double)));
    gpuErrchk(hipMemcpy(cv_d, cv, (*nov) * sizeof(double), hipMemcpyHostToDevice));

  

    double *max = new double(0);
    double *max_d;
    int temp_d = (*nov)-1;
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    for(int x = 0; x < siter;x++){
      gpuErrchk(hipMalloc( (void**) &max_d, sizeof(double)));
      updater<<<(*nov + 1024 - 1)/1024,1024>>>(rv_d,cv_d,adj_d,xadj_d,temp_d);
      gpuErrchk(hipPeekAtLastError());

      updaters<<<(*nov + 1024 - 1)/1024,1024>>>(rv_d,cv_d,tadj_d,txadj_d,temp_d);
      gpuErrchk(hipPeekAtLastError());

      updatere<<<(*nov + 1024 - 1)/1024,1024>>>(rv_d,cv_d,adj_d,xadj_d,max_d,temp_d);

      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipMemcpy(max, max_d, sizeof(double), hipMemcpyDeviceToHost));

      cout<<"iter "<< x <<" - error " <<*max<<endl;
      *max = 0;
    } 

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    gpuErrchk(hipFree(max_d));
    gpuErrchk(hipFree(xadj_d));
    gpuErrchk(hipFree(adj_d));
    gpuErrchk(hipFree(txadj_d));
    gpuErrchk(hipFree(tadj_d));
    gpuErrchk(hipFree(rv_d));
    gpuErrchk(hipFree(cv_d));
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU scale took: %f s\n", elapsedTime/1000);  
}

